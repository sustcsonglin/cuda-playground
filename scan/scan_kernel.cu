#include "hip/hip_runtime.h"
#include <torch/types.h>



__global__ void parallel_scan_kernel(float* d_out, const float* d_in, int B, int D, int L) {
    __shared__ float warp_sums[32];

    int tid = threadIdx.x;
    int laneId = tid & 31;
    int warpId = tid >> 5;

    int b = blockIdx.x;
    int d = blockIdx.y;

    int global_idx = ((b * D + d) * L) + tid;
    
    float value = d_in[global_idx];
    
    // Compute inclusive scan within warp
    for (int offset = 1; offset < 32; offset *= 2) {
        float up_value = __shfl_up_sync(0xffffffff, value, offset);
        if (laneId >= offset) {
            value += up_value;
        }
    }

    // The last thread of each warp writes its result to shared memory
    if (laneId == 31) {
        warp_sums[warpId] = value;
    }

    __syncthreads();

    // Use the first warp to compute the scan of the warp_sums
    if (warpId == 0) {
        float acc_value = warp_sums[laneId];
        
    
        for (int offset = 1; offset < 32; offset *= 2) {
            float up_value = __shfl_up_sync(0xffffffff, acc_value, offset);
            if (laneId >= offset) {
            acc_value += up_value;
            }
        }

        warp_sums[laneId] = acc_value;
    }

    __syncthreads();

    // Add the scanned sum of the previous warps to the current warp's result
    if (warpId != 0) {
        value += warp_sums[warpId - 1];
    }

    // Store result
    d_out[tid + ((b * D + d) * L)] = value;
}



void parallel_scan(torch::Tensor d_out, torch::Tensor d_in, int B, int D, int L) {
    const dim3 blockSize(1024, 1, 1);
    const dim3 gridSize(B, D, 1);
    
    parallel_scan_kernel<<<gridSize, blockSize>>>(d_out.data_ptr<float>(), d_in.data_ptr<float>(), B, D, L);
}



